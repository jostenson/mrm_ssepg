#include "hip/hip_runtime.h"
/*
  This file is part of the MRF_CUDA package (https://github.com/chixindebaoyu/MRF_CUDA)

  The MIT License (MIT)

  Copyright (c) 2019 Dong Wang and David Smith

  Permission is hereby granted, free of charge, to any person obtaining a # copy
  of this software and associated documentation files (the "Software"), to # deal
  in the Software without restriction, including without limitation the # rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or # sell
  coM_PIes of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included # in all
  coM_PIes or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS # OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL # THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING # FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS # IN THE
  SOFTWARE.
*/

// #include "functions.h"
#include <stdbool.h>
#include <stdlib.h>
#include <stdio.h>
#include <fcntl.h>
#include <errno.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <stdint.h>
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "float2math.h"

// Specify the sizes according to the divice
// int blocksize = 64;
// int gridsize = 128;

// kernels


__global__ void
dephase_gradients_rf_stage1(float2 * d_w_out, float2 * d_w, size_t natoms, int nstates)
{
    int ii, jj, idx;
    int from_idx, to_idx;

    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < 2 * natoms * nstates - 2;
        id += blockDim.x * gridDim.x)
    {

		ii = id / 2;
		jj = natoms * nstates - ii - 1;
        idx = (id + 1) % 2;

        from_idx = (1 - idx) * jj + idx * ii;
        to_idx = from_idx + 2 * idx - 1;

        d_w_out[3 * from_idx + 1 - idx] =
            d_w[3 * to_idx + 1 - idx];
    }
}

__global__ void
dephase_gradients_rf_stage2(float2 * d_w, size_t natoms, int nstates)
{
    int idx;

    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < natoms;
        id += blockDim.x * gridDim.x)
    {
        idx = id * 3 * nstates + 1;

        d_w[idx] = conj(d_w[idx - 1]);
        d_w[idx + 3 * nstates - 4] = make_float2(0.f);
    }
}

